#include "hip/hip_runtime.h"
/*
Group Members:
Joshua Taing
Max Mazal
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

#include "vector.h"
#include "config.h"

extern vector3* hVel;
extern vector3* hPos;
extern double* mass;

//Function that computes the pairwise accelerations. Effect is on the first argument.
__global__ void comp_PA(vector3* hPos, double* mass, vector3** accels){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    //int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUMENTITIES) {
		return;
	}
    
    //This part was just C+P'd from the original compute.c -- only change is
    //that it's not a for loop, since it should be looping in the for loop in nbody.c's main instead.
    for (int j = 0; j < NUMENTITIES; j++){
        if (i == j) {
            FILL_VECTOR(accels[i][j], 0, 0, 0);
        }
        else {
            vector3 d; //distance
            for (int k = 0; k < 3; k++){
                d[k] = hPos[i][k] - hPos[j][k];
            }
            double magnitude_sq = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];
            double mag = sqrt(magnitude_sq);
			double accelmag = -1 * GRAV_CONSTANT * mass[j] / magnitude_sq;
			FILL_VECTOR(accels[i][j], accelmag*d[0]/mag, accelmag*d[1]/mag, accelmag*d[2]/mag);
        }
    }
}

//Function to sum rows of the matrix, then update velocity/position.
__global__ void sum_update(vector3* hVel, vector3* hPos, vector3** accels){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUMENTITIES) {
		return;
	}

    vector3 accel_sum = {0, 0, 0};
    for (int j = 0; j < NUMENTITIES; j++) {
        for (int k = 0; k < 3; k++) {
            accel_sum[k] += accels[i][j][k];
        }
    }
    //Update velocity and postion.
    for (int k = 0; k < 3; k++) {
        hVel[i][k] += accel_sum[k] * INTERVAL;
        hPos[i][k] += hVel[i][k] * INTERVAL;
    }
}

//Host variables
extern vector3* hVel;
extern vector3* hPos;
extern double* mass;

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
//Use this function to call parallelized functions above
void compute() {
	dim3 blockDim(16, 16);
	dim3 gridDim((NUMENTITIES + blockDim.x - 1) / blockDim.x, (NUMENTITIES + blockDim.y - 1) / blockDim.y);

    vector3* d_hVel;
    vector3* d_hPos;
    vector3** d_accels;
    double* d_mass;

    hipMalloc((void**)&d_hVel, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**)&d_hPos, sizeof(vector3) * NUMENTITIES);
	hipMalloc((void**)&d_mass, sizeof(double) * NUMENTITIES);
	hipMalloc((void**)&d_accels, sizeof(vector3) * NUMENTITIES * NUMENTITIES);

    hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);

    comp_PA<<<gridDim, blockDim>>>(d_hPos, d_mass, d_accels);
    hipDeviceSynchronize();

    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) 
    //    printf("Error: %s\n", hipGetErrorString(err));

    sum_update<<<gridDim, blockDim>>>(d_hVel, d_hPos, d_accels);
    hipDeviceSynchronize();

    hipMemcpy(hPos, d_hPos, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hVel, d_hVel, sizeof(vector3)*NUMENTITIES, hipMemcpyDeviceToHost);
    //hipMemcpy(mass, d_mass, sizeof(double)*NUMENTITIES, hipMemcpyDeviceToHost);

    hipFree(d_hVel);
	hipFree(d_hPos);
	hipFree(d_mass);
	hipFree(d_accels);
}
