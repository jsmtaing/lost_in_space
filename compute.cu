#include "hip/hip_runtime.h"
/*
Group Members:
Joshua Taing
Max Mazal
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <stdio.h>


extern __shared__ double sharedAccels[];

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
__global__ void compute(vector3 *d_accels, vector3 *d_accel_sum, vector3 *d_hVel, vector3 *d_hPos, double *d_mass) {
	
}

