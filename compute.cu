#include "hip/hip_runtime.h"
/*
Group Members:
Joshua Taing
Max Mazal
*/

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"

extern __shared__ double sharedAccels[];

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
__global__ void compute(vector3 *d_accels, vector3 *d_accel_sum, vector3 *d_hVel, vector3 *d_hPos, double *d_mass) {
	//thread indices
	int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

	//block indices
	int blockRow = blockIdx.y;
	// int blockCol = blockIdx.x;
    
	//Max 12.3.23 12pm
	//declares shared memory arrays to store data to be shared among threads within the block
    __shared__ double sharedMass[BLOCK_SIZE];
    __shared__ vector3 sharedPos[BLOCK_SIZE];
    __shared__ vector3 sharedVel[BLOCK_SIZE];
    __shared__ vector3 sharedAccels[BLOCK_SIZE * BLOCK_SIZE];


	//Max 12.3.23 12pm
	//these arrays load each thread's mass, position, and velocity data into the shared memory (respectively)
	sharedMass[threadIdx.y] = d_mass[row];
    for (int k = 0; k < 3; k++)
    	sharedPos[threadIdx.y][k] = d_hPos[row][k];
    for (int k = 0; k < 3; k++)
    	sharedVel[threadIdx.y][k] = d_hVel[row][k];



	//Max 12.3.23 12pm
	//syncs the threads to ensure each block finished loading data into shared memory before procceeding with computation
	__syncthreads();

	//first compute the pairwise accelerations.  Effect is on the first argument.
	vector3 tempAccel = {0, 0, 0};
    for (int j = 0; j < BLOCK_SIZE; j++){
        vector3 distance;
        for (int k = 0; k < 3; k++)
			distance[k] = sharedPos[row][k] - sharedPos[j][k];

        double magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] + distance[2] * distance[2];
        double magnitude = sqrt(magnitude_sq);
        double accelmag = -1 * GRAV_CONSTANT * sharedMass[j] / magnitude_sq;

        for (int k = 0; k < 3; k++)
            tempAccel[k] += accelmag * distance[k] / magnitude;
   	}

	__syncthreads();

	//gets the sum of the rows of the matrix
	for (int k = 0; k < 3; k++)
        sharedAccels[threadIdx.y * BLOCK_SIZE + threadIdx.x][k] = tempAccel[k];

	__syncthreads();

	//computes pairwise accelerations
	vector3 accelSum = {0, 0, 0};
    for (int j = 0; j < BLOCK_SIZE; j++) {
        for (int k = 0; k < 3; k++)
            accelSum[k] += sharedAccels[threadIdx.y * BLOCK_SIZE + j][k];
    }

	//updates velocity and position
	 for (int k = 0; k < 3; k++) {
        sharedVel[threadIdx.y][k] += accelSum[k] * INTERVAL;
        sharedPos[threadIdx.y][k] += sharedVel[threadIdx.y][k] * INTERVAL;
    }

	__syncthreads();

    //copies data back to global memory
    for (int k = 0; k < 3; k++) {
        d_hVel[row][k] = sharedVel[threadIdx.y][k];
        d_hPos[row][k] = sharedPos[threadIdx.y][k];
    }

	vector3* values = (vector3*)malloc(sizeof(vector3) * NUMENTITIES * NUMENTITIES);
    vector3** accels = (vector3**)malloc(sizeof(vector3*) * NUMENTITIES);
    for (int i = 0; i < NUMENTITIES; i++)
        accels[i] = &values[i * NUMENTITIES];


	// //sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	// for (int i =0;i<NUMENTITIES;i++){
	// 	vector3 accel_sum={0,0,0};
	// 	for (int j=0;j<NUMENTITIES;j++){
	// 		for (int k=0;k<3;k++)
	// 			accel_sum[k]+=accels[i][j][k];
	// 	}
	// 	//compute the new velocity based on the acceleration and time interval
	// 	//compute the new position based on the velocity and time interval
	// 	for (int k=0;k<3;k++){
	// 		hVel[i][k]+=accel_sum[k]*INTERVAL;
	// 		hPos[i][k]+=hVel[i][k]*INTERVAL;
	// 	}
	// }

	for (int i = 0; i < NUMENTITIES; i++)
    	delete[] accels[i];

	delete[] accels;
	delete[] values;
}

